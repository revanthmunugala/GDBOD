#include "hip/hip_runtime.h"
#include "hySortOD_lib.h"

using namespace std;

int main(int argc, char **argv) 
{
    // Process command-line arguments
    int N;
    int DIM;
    int BIN;
    int MINSPLIT; // MINSPLIT = 0 defaults to naive strategy
    int NORMALIZE = 1; // Set to 1 to normalize datasets - Does not affect timeTrails
    char inputFname[500] = ""; // Dataset
    int APPROACH = 1; // 0 for Naive strategy and 1 for Tree strategy
    int TREE_SELECT = 3; // Optional parameter (default -3) 1 for simple tree, 2 for tree with optimized locality, 3 for tree with optimized locality and traversal, Use 0 for naive strategy
    
    if (argc != 9)
    {
        fprintf(stderr, "Please provide the following on the command line: N (number of lines in the file), dimensionality (number of coordinates per point/feature vector), BIN (Bin parameter), Min Split(Threshold), Normalize (0 or 1), dataset filename. ");
         return 0;
    }

    sscanf(argv[1], "%d", &N);
    sscanf(argv[2], "%d", &DIM);
    sscanf(argv[3], "%d", &BIN);
    sscanf(argv[4], "%d", &MINSPLIT);
    sscanf(argv[5], "%d", &NORMALIZE);
    strcpy(inputFname, argv[6]);
    sscanf(argv[7], "%d", &APPROACH);
    sscanf(argv[8], "%d", &TREE_SELECT);
    
    if (N < 1 || DIM < 1 || BIN < 1 || MINSPLIT<0 || NORMALIZE > 1 || NORMALIZE < 0 || APPROACH < 0 || APPROACH > 1 || TREE_SELECT < 0 || TREE_SELECT > 3) {
        cout << "\nOne of the following are invalid: N, DIM, BIN , NORMALIZE, APPROACH, TREE_SELECT " << endl;
        return 0;
    } else {

        string currentTreeSelect = "NONE";

        if(APPROACH == 1)
        {
            currentTreeSelect = treeSelect[TREE_SELECT-1];
        }

        cout << "\nNumber of lines (N): " << N << " Dimensionality: " << DIM << " BIN Size: "<< BIN
               << " MinSplit: " << MINSPLIT << " Normalize: " <<NORMALIZE << " Filename: " << inputFname << " Approach: " << approach[APPROACH]
               << " Selected tree: " << currentTreeSelect << endl;

    }

     // allocate memory for dataset
        // pointer to entire dataset
    double *h_dataset = (double *) malloc(sizeof(double) * N * DIM);

    int ret = importDataset(inputFname, N, h_dataset, DIM);

    if (ret == 1) {
        return 0;
    }
    
    if (NORMALIZE == 1) {
        normalizeDataset(h_dataset, N, DIM);
    }

    hipEvent_t totalTimeStart, totalTimeStop;
    hipEventCreate(&totalTimeStart);
    hipEventCreate(&totalTimeStop);

    hipEventRecord(totalTimeStart);

    int blockDim = 32;

    while (blockDim <= DIM) {
        blockDim += 32;
    }

    if (blockDim > 1024) {
        printf("\nMax allowed dimensions is 1024");
        return 0;
    }

    int totalElements = N * DIM;

    int totalElementsPerBlock = blockDim / DIM;

    dim3 dimGrid(ceil((float) N / (float) totalElementsPerBlock), 1, 1);
    dim3 dimBlock(blockDim, 1, 1);

    int k = findK(BIN);

    int dimPerBlock = floor((double)(sizeof(MY_DATATYPE)*8)/(double)k);
  
    int encodeBlockSize = ceil((double) DIM / (double) dimPerBlock);

    MY_DATATYPE *d_hypercube = nullptr;

    double *d_dataset = nullptr;

    MY_DATATYPE*h_hypercube = (MY_DATATYPE *) calloc(encodeBlockSize * N, sizeof(MY_DATATYPE));

    hipMalloc((void **) &d_hypercube, (sizeof(MY_DATATYPE) * N * encodeBlockSize));

    hipMalloc((void **) &d_dataset, sizeof(double) * totalElements);

    hipMemcpy(d_dataset, h_dataset, sizeof(double) * totalElements,
               hipMemcpyHostToDevice);

    hipMemcpy(d_hypercube, h_hypercube, (sizeof(MY_DATATYPE) * N * encodeBlockSize), hipMemcpyHostToDevice);

    // Record time
    hipEvent_t createHypercubeStart, createHypercubeStop;
    hipEventCreate(&createHypercubeStart);
    hipEventCreate(&createHypercubeStop);

    hipEventRecord(createHypercubeStart);

    createHypercube<<<dimGrid, dimBlock,
    sizeof(int) * totalElementsPerBlock * DIM>>>(
            d_hypercube, d_dataset, N, DIM, BIN, encodeBlockSize, k);

    hipDeviceSynchronize();

    hipEventRecord(createHypercubeStop);
    hipEventSynchronize(createHypercubeStop);

    // Calculate elapsed time
    float createHypercubeTime = 0;
    hipEventElapsedTime(&createHypercubeTime, createHypercubeStart, createHypercubeStop);

    hipMemcpy(h_hypercube, d_hypercube, (sizeof(MY_DATATYPE) * N * encodeBlockSize), hipMemcpyDeviceToHost);

    // Improve memory util
    hipFree(d_dataset);
    hipFree(d_hypercube);

    map <vector<MY_DATATYPE>, vector<int>> h_hypercube_mapper;

    for (int i = 0; i + encodeBlockSize <= N * encodeBlockSize; i = i + encodeBlockSize) {

        vector<MY_DATATYPE> h_hypercube_key(h_hypercube + i, h_hypercube + i + encodeBlockSize);

        if (h_hypercube_mapper.find(h_hypercube_key) == h_hypercube_mapper.end()) {
            vector<int> h_hypercube_instance;
            h_hypercube_mapper[h_hypercube_key] = h_hypercube_instance;
        }

        h_hypercube_mapper[h_hypercube_key].push_back(i / encodeBlockSize);

    }

    int distinctHypercubeCount = 0;
    MY_DATATYPE* h_hypercubeDistinct = nullptr;

    int *h_instancesCount = nullptr;

    distinctHypercubeCount = h_hypercube_mapper.size();

    h_hypercubeDistinct = (MY_DATATYPE *) malloc(sizeof(MY_DATATYPE) * distinctHypercubeCount * encodeBlockSize);

    h_instancesCount = (int *) malloc(sizeof(int) * distinctHypercubeCount);
    
    map < vector <MY_DATATYPE> , vector < int > > ::iterator itr;

    int hypercubePos = 0;
    for (itr = h_hypercube_mapper.begin(); itr != h_hypercube_mapper.end(); itr++) {
        memcpy(h_hypercubeDistinct + hypercubePos, itr->first.data(), sizeof(MY_DATATYPE) * itr->first.size());
        h_instancesCount[hypercubePos / itr->first.size()] = itr->second.size();
        hypercubePos += itr->first.size();
    }

    // Building Array of Hypercubes

    printf("Distinct Hypercube Count: %d\n", distinctHypercubeCount);

    totalElements = distinctHypercubeCount * DIM;

    MY_DATATYPE *d_hypercubeDistinct;

    int *d_hypercubeArray;

    int *h_hypercubeArray = (int *) malloc(sizeof(int) * totalElements);

    hipMalloc((void **) &d_hypercubeDistinct,
               sizeof(MY_DATATYPE) * distinctHypercubeCount * encodeBlockSize);

    hipMalloc((void **) &d_hypercubeArray, sizeof(int) * totalElements);

    hipMemcpy(d_hypercubeDistinct, h_hypercubeDistinct, sizeof(MY_DATATYPE) * distinctHypercubeCount * encodeBlockSize,
               hipMemcpyHostToDevice);

    dimGrid.x = ceil((float) totalElements / (float) blockDim);

    printf("Grid - %.0f , Block - %d\n", ceil((float) totalElements / (float) blockDim), blockDim);

    // Record time here
    hipEvent_t buildHypercubeArrayStart, buildHypercubeArrayStop;
    hipEventCreate(&buildHypercubeArrayStart);
    hipEventCreate(&buildHypercubeArrayStop);

    hipEventRecord(buildHypercubeArrayStart);

    buildHypercubeArray<<<dimGrid, dimBlock>>>(
            d_hypercubeDistinct, d_hypercubeArray, distinctHypercubeCount, DIM, encodeBlockSize, k);

    hipDeviceSynchronize();

    hipEventRecord(buildHypercubeArrayStop);
    hipEventSynchronize(buildHypercubeArrayStop);

    float buildHypercubeArrayTime = 0;
    hipEventElapsedTime(&buildHypercubeArrayTime, buildHypercubeArrayStart, buildHypercubeArrayStop);

    int *h_neighborhoodDensity = (int *) calloc(distinctHypercubeCount, sizeof(int));
     
    float neighborhoodDensityTime;

    if (APPROACH == 0) 
    {   
        puts("Using naive approach");
        neighborhoodDensityTime = naiveStrategy(d_hypercubeArray, h_neighborhoodDensity, h_instancesCount, distinctHypercubeCount, BIN, DIM);
        
    } else 
    {
        if(TREE_SELECT == 1)
        {
            neighborhoodDensityTime = simpleTreeStrategy(h_hypercubeArray, d_hypercubeArray, h_neighborhoodDensity,h_instancesCount, distinctHypercubeCount, DIM, MINSPLIT);

        }else if(TREE_SELECT == 2)
        {
            neighborhoodDensityTime = localityOptimTreeStrategy(h_hypercubeArray, d_hypercubeArray, h_neighborhoodDensity,h_instancesCount, distinctHypercubeCount, DIM, MINSPLIT);

        }else
        {
            neighborhoodDensityTime = finalOptimTreeStrategy(h_hypercubeArray, d_hypercubeArray, h_neighborhoodDensity,h_instancesCount, distinctHypercubeCount, DIM, MINSPLIT);
        }
       
    }

    int maxNeighborhoodDensity = INT_MIN;

    for (int i = 0; i < distinctHypercubeCount; i++) 
    {   
        
        if (i < 50)
        printf("Index - %d - %d\n",i,h_neighborhoodDensity[i]);
        
        if (h_neighborhoodDensity[i] > maxNeighborhoodDensity) {
            maxNeighborhoodDensity = h_neighborhoodDensity[i];
        }
    }

    // Calculate Outlier Score 
    float *h_outlierScore =
            (float *) malloc(sizeof(float) * N);

    calculateOutlierScore(
            h_outlierScore, h_neighborhoodDensity, h_hypercube_mapper, N,
            maxNeighborhoodDensity);
    clock_t end = clock();
    
    hipDeviceSynchronize();
    hipEventRecord(totalTimeStop);
    hipEventSynchronize(totalTimeStop);

    float totalTime = 0;
    hipEventElapsedTime(&totalTime, totalTimeStart, totalTimeStop);

    printf("============TIME RESULTS================\n");

    printf("Total time for execution is %f sec \n", (totalTime / 1000));

    printf("Total time for building hypercube is %f sec \n", (buildHypercubeArrayTime / 1000));

    printf("Time for neighborhood density is %f sec \n", (neighborhoodDensityTime / (1000)));

    hipError_t error = hipGetLastError();

    if (error != hipSuccess) {
        printf("%s \n", hipGetErrorString(error));
    }

    // free dataset
    free(h_dataset);

    return 0;
}
