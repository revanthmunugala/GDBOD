#include "hip/hip_runtime.h"
#include "hySortOD_lib.h"

using namespace std;

int main(int argc, char **argv) 
{
    
    // Process command-line arguments
    int N;
    int DIM;
    int BIN;
    int MINSPLIT; // MINSPLIT = 0 defaults to naive strategy
    int NORMALIZE = 1; // Set to 1 to normalize datasets - Does not affect timeTrails
    char inputFname[500] = ""; // Dataset
    int APPROACH = 1; // 0 for Naive strategy and 1 for Tree strategy
    int TREE_SELECT = 3; // Optional parameter (default -3) 1 for simple tree, 2 for tree with optimized locality, 3 for tree with optimized locality and traversal, Use 0 for naive strategy
    
    if (argc != 9)
    {
        fprintf(stderr, "Please provide the following on the command line: N (number of lines in the file), dimensionality (number of coordinates per point/feature vector), BIN (Bin parameter), Min Split(Threshold), Normalize (0 or 1), dataset filename. ");
         return 0;
    }

    sscanf(argv[1], "%d", &N);
    sscanf(argv[2], "%d", &DIM);
    sscanf(argv[3], "%d", &BIN);
    sscanf(argv[4], "%d", &MINSPLIT);
    sscanf(argv[5], "%d", &NORMALIZE);
    strcpy(inputFname, argv[6]);
    sscanf(argv[7], "%d", &APPROACH);
    sscanf(argv[8], "%d", &TREE_SELECT);
    
    if (N < 1 || DIM < 1 || BIN < 1 || MINSPLIT<0 || NORMALIZE > 1 || NORMALIZE < 0 || APPROACH < 0 || APPROACH > 1 || TREE_SELECT < 0 || TREE_SELECT > 3) {
        cout << "\nOne of the following are invalid: N, DIM, BIN , NORMALIZE, APPROACH, TREE_SELECT " << endl;
        return 0;
    } else {

        string currentTreeSelect = "NONE";

        if(APPROACH == 1)
        {
            currentTreeSelect = treeSelect[TREE_SELECT-1];
        }

        cout << "\nNumber of lines (N): " << N << " Dimensionality: " << DIM << " BIN Size: "<< BIN
               << " MinSplit: " << MINSPLIT << " Normalize: " <<NORMALIZE << " Filename: " << inputFname << " Approach: " << approach[APPROACH]
               << " Selected tree: " << currentTreeSelect << endl;

    }


     // allocate memory for dataset
        // pointer to entire dataset
    double *h_dataset = (double *) malloc(sizeof(double) * N * DIM);

    int ret = importDataset(inputFname, N, h_dataset, DIM);

    if (ret == 1) {
        return 0;
    }
    
    if (NORMALIZE == 1) {
        normalizeDataset(h_dataset, N, DIM);
    }

    hipEvent_t totalTimeStart, totalTimeStop;
    hipEventCreate(&totalTimeStart);
    hipEventCreate(&totalTimeStop);

    hipEventRecord(totalTimeStart);

    int blockDim = 32;

    while (blockDim <= DIM) {
        blockDim += 32;
    }

    if (blockDim > 1024) {
        printf("\nMax allowed dimensions is 1024");
        return 0;
    }

    int totalElements = N * DIM;

    int totalElementsPerBlock = blockDim / DIM;

    dim3 dimGrid(ceil((float) N / (float) totalElementsPerBlock), 1, 1);
    dim3 dimBlock(blockDim, 1, 1);

    int k = findK(BIN);

    int dimPerBlock = floor((double)(sizeof(MY_DATATYPE)*8)/(double)k);
  
    int encodeBlockSize = ceil((double) DIM / (double) dimPerBlock);

    MY_DATATYPE *d_hypercube = nullptr;

    double *d_dataset = nullptr;

    printf("Grid - %.0f , Block - %d\n", ceil((float) N / (float) totalElementsPerBlock), blockDim);

    MY_DATATYPE*h_hypercube = (MY_DATATYPE *) calloc(encodeBlockSize * N, sizeof(MY_DATATYPE));

    hipMalloc((void **) &d_hypercube, (sizeof(MY_DATATYPE) * N * encodeBlockSize));

    hipMalloc((void **) &d_dataset, sizeof(double) * totalElements);

    hipMemcpy(d_dataset, h_dataset, sizeof(double) * totalElements,
               hipMemcpyHostToDevice);

    hipMemcpy(d_hypercube, h_hypercube, (sizeof(MY_DATATYPE) * N * encodeBlockSize), hipMemcpyHostToDevice);

    // Record time
    hipEvent_t createHypercubeStart, createHypercubeStop;
    hipEventCreate(&createHypercubeStart);
    hipEventCreate(&createHypercubeStop);

    hipEventRecord(createHypercubeStart);

    createHypercube<<<dimGrid, dimBlock,
    sizeof(int) * totalElementsPerBlock * DIM>>>(
            d_hypercube, d_dataset, N, DIM, BIN, encodeBlockSize, k);

    hipDeviceSynchronize();

    hipEventRecord(createHypercubeStop);
    hipEventSynchronize(createHypercubeStop);

    // Calculate elapsed time
    float createHypercubeTime = 0;
    hipEventElapsedTime(&createHypercubeTime, createHypercubeStart, createHypercubeStop);

    hipMemcpy(h_hypercube, d_hypercube, (sizeof(MY_DATATYPE) * N * encodeBlockSize), hipMemcpyDeviceToHost);

    // Improve memory util
    hipFree(d_dataset);
    hipFree(d_hypercube);

    map <vector<MY_DATATYPE>, vector<int>> h_hypercube_mapper;

    for (int i = 0; i + encodeBlockSize <= N * encodeBlockSize; i = i + encodeBlockSize) {

        vector<MY_DATATYPE> h_hypercube_key(h_hypercube + i, h_hypercube + i + encodeBlockSize);

        if (h_hypercube_mapper.find(h_hypercube_key) == h_hypercube_mapper.end()) {
            vector<int> h_hypercube_instance;
            h_hypercube_mapper[h_hypercube_key] = h_hypercube_instance;
        }

        h_hypercube_mapper[h_hypercube_key].push_back(i / encodeBlockSize);

    }

    int distinctHypercubeCount = 0;
    MY_DATATYPE* h_hypercubeDistinct = nullptr;

    int *h_instancesCount = nullptr;


    distinctHypercubeCount = h_hypercube_mapper.size();

    h_hypercubeDistinct = (MY_DATATYPE *) malloc(sizeof(MY_DATATYPE) * distinctHypercubeCount * encodeBlockSize);

    h_instancesCount = (int *) malloc(sizeof(int) * distinctHypercubeCount);
    
    map < vector <MY_DATATYPE> , vector < int > > ::iterator itr;

    int hypercubePos = 0;
    for (itr = h_hypercube_mapper.begin(); itr != h_hypercube_mapper.end(); itr++) {
        memcpy(h_hypercubeDistinct + hypercubePos, itr->first.data(), sizeof(MY_DATATYPE) * itr->first.size());
        h_instancesCount[hypercubePos / itr->first.size()] = itr->second.size();
        hypercubePos += itr->first.size();
    }

    // Building Array of Hypercubes

    printf("Distinct Hypercube Count: %d\n", distinctHypercubeCount);

    totalElements = distinctHypercubeCount * DIM;

    MY_DATATYPE *d_hypercubeDistinct;

    int *d_hypercubeArray;

    int *h_hypercubeArray = (int *) malloc(sizeof(int) * totalElements);

    hipMalloc((void **) &d_hypercubeDistinct,
               sizeof(MY_DATATYPE) * distinctHypercubeCount * encodeBlockSize);

    hipMalloc((void **) &d_hypercubeArray, sizeof(int) * totalElements);

    hipMemcpy(d_hypercubeDistinct, h_hypercubeDistinct, sizeof(MY_DATATYPE) * distinctHypercubeCount * encodeBlockSize,
               hipMemcpyHostToDevice);

    dimGrid.x = ceil((float) totalElements / (float) blockDim);

    printf("Grid - %.0f , Block - %d\n", ceil((float) totalElements / (float) blockDim), blockDim);

    // Record time here
    hipEvent_t buildHypercubeArrayStart, buildHypercubeArrayStop;
    hipEventCreate(&buildHypercubeArrayStart);
    hipEventCreate(&buildHypercubeArrayStop);

    hipEventRecord(buildHypercubeArrayStart);

    buildHypercubeArray<<<dimGrid, dimBlock>>>(
            d_hypercubeDistinct, d_hypercubeArray, distinctHypercubeCount, DIM, encodeBlockSize, k);

    hipDeviceSynchronize();

    hipEventRecord(buildHypercubeArrayStop);
    hipEventSynchronize(buildHypercubeArrayStop);

    float buildHypercubeArrayTime = 0;
    hipEventElapsedTime(&buildHypercubeArrayTime, buildHypercubeArrayStart, buildHypercubeArrayStop);

    int *h_neighborhoodDensity = (int *) calloc(distinctHypercubeCount, sizeof(int));
    int *d_neighborhoodDensity = NULL;
    int *d_instancesCount = NULL;

    hipMalloc((void **) &d_neighborhoodDensity,
               sizeof(int) * distinctHypercubeCount);

    hipMalloc((void **) &d_instancesCount, sizeof(int) * distinctHypercubeCount);

    hipMemcpy(d_neighborhoodDensity, h_neighborhoodDensity,
               sizeof(int) * distinctHypercubeCount, hipMemcpyHostToDevice);

    hipMemcpy(d_instancesCount, h_instancesCount,
               sizeof(int) * distinctHypercubeCount, hipMemcpyHostToDevice);

    hipEvent_t neighborhoodDensityStart, neighborhoodDensityStop, buildTreeStart, buildTreeStop;
    hipEvent_t buildSuperOptimTreeStart, buildSuperOptimTreeStop;
    hipEvent_t buildOptimTreeStart, buildOptimTreeStop;
    float neighborhoodDensityTime = 0, treeTime = 0, superOptimTreeTime = 0, optimTreeTime = 0;


    if (MINSPLIT == 0) {
        dimBlock.x = 256;
        int SPLIT = BIN;
        dimGrid.x = ceil((float) (distinctHypercubeCount * SPLIT) / (float) (dimBlock.x));

        hipEventCreate(&neighborhoodDensityStart);
        hipEventCreate(&neighborhoodDensityStop);
        hipEventRecord(neighborhoodDensityStart);

        naiveNeighborhoodDensity<<<dimGrid, dimBlock>>>(d_neighborhoodDensity, d_instancesCount, DIM,
                                                        distinctHypercubeCount, d_hypercubeArray, SPLIT);

        hipDeviceSynchronize();
        hipEventRecord(neighborhoodDensityStop);
        hipEventSynchronize(neighborhoodDensityStop);
        hipEventElapsedTime(&neighborhoodDensityTime, neighborhoodDensityStart, neighborhoodDensityStop);

    } else {

        hipMemcpy(h_hypercubeArray, d_hypercubeArray, sizeof(int) * totalElements,
                   hipMemcpyDeviceToHost);

        // Build a linear tree

        treeNode *h_linearTree = NULL;
        int linearTreeCount = 0;
        int curDim = 0;

        int *h_childCount = (int *) calloc((DIM + 1), sizeof(int));
        int *h_dimStart = (int *) calloc((DIM + 1), sizeof(int));

        appendNode(&h_linearTree, 0, distinctHypercubeCount - 1, NONE, NONE,
                   &linearTreeCount);

        hipEventCreate(&buildTreeStart);
        hipEventCreate(&buildTreeStop);

        hipEventRecord(buildTreeStart);

        buildLinearTree(h_hypercubeArray, &h_linearTree, h_childCount, h_dimStart,
                        &linearTreeCount, curDim, distinctHypercubeCount, MINSPLIT, DIM);

        hipDeviceSynchronize();

        hipEventRecord(buildTreeStop);
        hipEventSynchronize(buildTreeStop);

        hipEventElapsedTime(&treeTime, buildTreeStart, buildTreeStop);

        printf("Tree Node Count: %d\n", linearTreeCount);

        // Build optimized linear tree with better cache hits
        // Build optimized linear tree here
        // Allocate memory for optim tree

        printf("Optim tree build start\n");

        treeNode *h_optimizedLinearTree =
                (treeNode *) malloc(sizeof(treeNode) * linearTreeCount);


        int *h_dimNodes = (int *) malloc(sizeof(int) * h_childCount[0]);

        hipEventCreate(&buildOptimTreeStart);
        hipEventCreate(&buildOptimTreeStop);

        hipEventRecord(buildOptimTreeStart);

        buildOptimizedLinearTree(h_linearTree, h_optimizedLinearTree, h_dimNodes);

        hipDeviceSynchronize();

        hipEventRecord(buildOptimTreeStop);
        hipEventSynchronize(buildOptimTreeStop);

        hipEventElapsedTime(&optimTreeTime, buildOptimTreeStart, buildOptimTreeStop);

        puts("Optimized tree build done");

        optimTreeNode *h_superOptimTree = (optimTreeNode *) malloc(sizeof(optimTreeNode) * linearTreeCount);

        hipEventCreate(&buildSuperOptimTreeStart);
        hipEventCreate(&buildSuperOptimTreeStop);

        hipEventRecord(buildSuperOptimTreeStart);

        buildSuperOptimTree(h_optimizedLinearTree, h_superOptimTree);

        hipDeviceSynchronize();

        hipEventRecord(buildSuperOptimTreeStop);
        hipEventSynchronize(buildSuperOptimTreeStop);

        hipEventElapsedTime(&superOptimTreeTime, buildSuperOptimTreeStart, buildSuperOptimTreeStop);

        puts("Super Optimized Tree build done!!");

        // Count neighborhood density of hypercubes

        h_dimStart[DIM] = h_childCount[DIM - 2] + h_dimStart[DIM - 1];

        optimTreeNode *d_linearTree = NULL;
        int *d_dimStart = NULL;
        int *d_childCount = NULL;

        //Adding new code below

        int *d_dimNodes = NULL;

        hipMalloc((void **) &d_dimNodes, sizeof(int) * h_childCount[0]);

        hipMemcpy(d_dimNodes, h_dimNodes,
                   sizeof(int) * h_childCount[0], hipMemcpyHostToDevice);


        // Check below line again

        hipMalloc((void **) &d_linearTree, sizeof(optimTreeNode) * linearTreeCount);

        hipMalloc((void **) &d_dimStart, sizeof(int) * (DIM + 1));

        hipMalloc((void **) &d_childCount, sizeof(int) * (DIM + 1));


        hipMemcpy(d_linearTree, h_superOptimTree, sizeof(optimTreeNode) * linearTreeCount,
                   hipMemcpyHostToDevice);


        hipMemcpy(d_dimStart, h_dimStart, sizeof(int) * (DIM + 1),
                   hipMemcpyHostToDevice);

        hipMemcpy(d_childCount, h_childCount, sizeof(int) * (DIM + 1),
                   hipMemcpyHostToDevice);


        dimBlock.x = 256;
        if (h_childCount[0] == 0) {
            dimGrid.x = ceil((float) (distinctHypercubeCount) / (float) (dimBlock.x));
        } else {
            dimGrid.x = ceil((float) (distinctHypercubeCount * h_childCount[0]) / (float) (dimBlock.x));
        }
        printf("Grid - %d  , Block - %d\n", dimGrid.x, dimBlock.x);
        printf("Starting to process %d hypercubes\n", distinctHypercubeCount);

        // Record time here

        hipEventCreate(&neighborhoodDensityStart);
        hipEventCreate(&neighborhoodDensityStop);

        hipEventRecord(neighborhoodDensityStart);

        optimNeighborhoodDensity<<<dimGrid, dimBlock>>>(
                d_neighborhoodDensity, d_instancesCount, d_linearTree, d_hypercubeArray,
                d_childCount, DIM, distinctHypercubeCount, d_dimNodes);

        hipDeviceSynchronize();

        hipEventRecord(neighborhoodDensityStop);
        hipEventSynchronize(neighborhoodDensityStop);

        hipEventElapsedTime(&neighborhoodDensityTime, neighborhoodDensityStart, neighborhoodDensityStop);

    }

    hipMemcpy(h_neighborhoodDensity, d_neighborhoodDensity,
               sizeof(int) * distinctHypercubeCount, hipMemcpyDeviceToHost);

    int maxNeighborhoodDensity = INT_MIN;

    for (int i = 0; i < distinctHypercubeCount; i++) {
        if (i < 50)
        printf("Index - %d - %d\n",i,h_neighborhoodDensity[i]);
        if (h_neighborhoodDensity[i] > maxNeighborhoodDensity) {
            maxNeighborhoodDensity = h_neighborhoodDensity[i];
        }
    }

    
    // Calculate Outlier Score

    float *h_outlierScore =
            (float *) malloc(sizeof(float) * N);

    calculateOutlierScore(
            h_outlierScore, h_neighborhoodDensity, h_hypercube_mapper, N,
            maxNeighborhoodDensity);
    clock_t end = clock();
    
   
    
    hipDeviceSynchronize();
    hipEventRecord(totalTimeStop);
    hipEventSynchronize(totalTimeStop);

    float totalTime = 0;
    hipEventElapsedTime(&totalTime, totalTimeStart, totalTimeStop);

    printf("============TIME RESULTS================\n");

    printf("Total time for execution is %f sec \n", (totalTime / 1000));

    printf("Total time for building hypercube is %f sec \n", (buildHypercubeArrayTime / 1000));

    //printf("Total time to sort is %f sec \n",(sortTime/1000));

    printf("Time for building linear Tree is %f sec \n", (treeTime / 1000));

    printf("Time for building optimized linear Tree is %f sec \n", (optimTreeTime / 1000));

    printf("Time for building super optimized linear Tree is %f sec \n", (superOptimTreeTime / 1000));

    printf("Time for neighborhood density is %f sec \n", (neighborhoodDensityTime / (1000)));

    hipError_t error = hipGetLastError();

    if (error != hipSuccess) {
        printf("%s \n", hipGetErrorString(error));
    }

    // free dataset
    free(h_dataset);

    return 0;
}
